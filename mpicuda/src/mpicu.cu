// --- CSCS (Swiss National Supercomputing Center) ---


#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C"
void set_gpu(int dev)
{
  hipSetDevice(dev);
}

extern "C"
void get_gpu_info(char *gpu_string, int dev)
{
  struct hipDeviceProp_t dprop;
  hipGetDeviceProperties(&dprop, dev);
  strcpy(gpu_string,dprop.name);
}





extern "C"
void get_more_gpu_info(int dev)
{
  int driverVersion = 0, runtimeVersion = 0;
  struct hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
  hipDriverGetVersion(&driverVersion);
  hipRuntimeGetVersion(&runtimeVersion);
  printf("  CUDA Driver Version / Runtime Version     %d.%d / %d.%d\n", driverVersion/1000, (driverVersion%100)/10, runtimeVersion/1000, (runtimeVersion%100)/10);
  printf("  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);
  
}





// Add two arrays on the device
__global__ void gpu_kernel(int *d_a1, int *d_a2, int *d_a3, int N) {
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < N)
    d_a3[idx] = d_a1[idx] + d_a2[idx];
  d_a1[idx] = idx ; // dummy
}







//#define SIZE 12
extern "C"
void run_gpu_kernel(int SIZE) {

  int i; 
  int a1[SIZE], a2[SIZE], a3[SIZE];  // Host arrays
  int *d_a1, *d_a2, *d_a3;           // Device arrays
  
  // Initalize the host input arrays
  for(i = 0; i < SIZE; i++) {
    a1[i] = i;
    a2[i] = 100*i;
  }
  
  // Allocate the device arrays and copy data over to the device
  hipMalloc((void**) &d_a1, sizeof(int)*SIZE);
  hipMalloc((void**) &d_a2, sizeof(int)*SIZE);
  hipMalloc((void**) &d_a3, sizeof(int)*SIZE);  
  hipMemcpy(d_a1, a1, sizeof(int)*SIZE, hipMemcpyHostToDevice);
  hipMemcpy(d_a2, a2, sizeof(int)*SIZE, hipMemcpyHostToDevice);

  // Zero out results
  hipMemset(d_a3, 0, sizeof(int)*SIZE);
  
  gpu_kernel<<<3, 4>>>(d_a1, d_a2, d_a3, SIZE);
  
  hipMemcpy(a3, d_a3, sizeof(int)*SIZE, hipMemcpyDeviceToHost);
  
  printf("%d %d %d\n", 0, SIZE/2, SIZE-1);
  printf("%d ", a3[0]);
  printf("%d ", a3[SIZE/2]);
  printf("%d ", a3[SIZE-1]);
  printf("\n");

  hipFree(d_a1);
  hipFree(d_a2);
  hipFree(d_a3);
}
